#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cpu_anim.h"
#include "book.h"
#include <stdio.h>

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

texture<float> texConstSrc;
texture<float> texIn;
texture<float> texOut;

__global__ void copy_const_kernel(float *iptr)
{
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c = tex1Dfetch(texConstSrc, offset);
	if (c != 0)
		iptr[offset] = c;
}

__global__ void blend_kernel(float *dst, bool distOut)
{
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.x * blockIdx.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (x == 0)
		left++;
	if (x == DIM - 1)
		right--;

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (y == 0)
		top += DIM;
	if (y == DIM - 1)
		bottom -= DIM;

	float t, l, c, r, b;
	if (distOut)
	{
		t = tex1Dfetch(texIn, top);
		l = tex1Dfetch(texIn, left);
		c= tex1Dfetch(texIn, offset);
		r = tex1Dfetch(texIn, right);
		b = tex1Dfetch(texIn, bottom);
	}
	else
	{
		t = tex1Dfetch(texOut, top);
		l = tex1Dfetch(texOut, left);
		c = tex1Dfetch(texOut, offset);
		r = tex1Dfetch(texOut, right);
		b = tex1Dfetch(texOut, bottom);
	}
	dst[offset] = c + SPEED * (t + b + l + r - 4 * c);
}

// global needed by the update routine
struct DataBlock
{
	unsigned char *output_bitmap;
	float *dev_inSrc;
	float *dev_outSrc;
	float *dev_constSrc;
	CPUAnimBitmap *bitmap;
	hipEvent_t start, stop;
	float totalTime;
	float frames;
};

void anim_gpu(DataBlock *d, int ticks)
{
	hipEventRecord(d->start, 0);
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	CPUAnimBitmap *bitmap = d->bitmap;

	// since tex is global and bound, we have to use a flag to
	// select which is in/out per iteration
	volatile bool dstOut = true;
	for (int i = 0; i < 90; i++)
	{
		float *in, *out;
		if (dstOut)
		{
			in = d->dev_inSrc;
			out = d->dev_outSrc;
		}
		else
		{
			out = d->dev_inSrc;
			in = d->dev_outSrc;
		}
		copy_const_kernel<<<blocks, threads>>>(in);
		blend_kernel<<<blocks, threads>>>(out, dstOut);
		dstOut = !dstOut;
	}
	float_to_color<<<blocks, threads>>>(d->output_bitmap, d->dev_inSrc);
	hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);
	hipEventRecord(d->stop, 0);
	hipEventSynchronize(d->stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, d->start, d->stop);

	d->totalTime += elapsedTime;
	++d->frames;
	printf("Average Time per frame: %3.1f ms\n", d->totalTime / d->frames);
}

void anim_exit(DataBlock *d)
{
	hipUnbindTexture(texIn);
	hipUnbindTexture(texOut);
	hipUnbindTexture(texConstSrc);
	hipFree(d->dev_inSrc);
	hipFree(d->dev_outSrc);
	hipFree(d->dev_constSrc);
	hipEventDestroy(d->start);
	hipEventDestroy(d->stop);
}

int main()
{
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;
	hipEventCreate(&data.start);
	hipEventCreate(&data.stop);
	hipMalloc((void**)&data.output_bitmap, bitmap.image_size());

	// assume float == 4 chars in size(i.e, rgba)
	hipMalloc((void**)&data.dev_inSrc, bitmap.image_size());
	hipMalloc((void**)&data.dev_outSrc, bitmap.image_size());
	hipMalloc((void**)&data.dev_constSrc, bitmap.image_size());
	hipBindTexture(NULL, texConstSrc, data.dev_constSrc, bitmap.image_size());
	hipBindTexture(NULL, texIn, data.dev_inSrc, bitmap.image_size());
	hipBindTexture(NULL, texOut, data.dev_outSrc, bitmap.image_size());

	float *temp = (float*)malloc(bitmap.image_size());
	for (int i = 0; i < DIM * DIM; i++)
	{
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if ((x>300) && (x<600) && (y>310) && (y<601))
			temp[i] = MAX_TEMP;
	}
	temp[DIM*100+100] = (MAX_TEMP + MIN_TEMP)/2;
	temp[DIM*700+100] = MIN_TEMP;
	temp[DIM*300+300] = MIN_TEMP;
	temp[DIM*200+700] = MIN_TEMP;
	for (int y=800; y<900; y++) 
	{
		for (int x=400; x<500; x++) 
		{
			temp[x+y*DIM] = MIN_TEMP;
		}
	}
	hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
	for (int y = 800; y < DIM; y++)
	{
		for (int x = 0; x < 200; x++)
		{
			temp[x + y * DIM] = MAX_TEMP;
		}
	}
	hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
	free(temp);
	bitmap.anim_and_exit((void (*)(void *, int))anim_gpu, (void (*)(void*))anim_exit);
}